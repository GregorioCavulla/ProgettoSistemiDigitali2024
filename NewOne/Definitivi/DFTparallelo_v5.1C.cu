//Shared Memory

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <stdint.h>
#include <sys/stat.h>
#include <hip/hip_runtime.h>

#define PI 3.14159265358979323846
#define BLOCK_SIZE 256

// Struttura per rappresentare un numero complesso con allineamento
typedef struct __align__(8) {
    float real;
    float imag;
} Complesso;


// Funzione per creare una stringa di timestamp
void createTimestamp(char *buffer, size_t size) {
    time_t now = time(NULL);
    struct tm *t = localtime(&now);
    strftime(buffer, size, "%Y%m%d_%H%M%S", t);
}

// Funzione per leggere l'intestazione di un file .wav e determinare la lunghezza
int getWavFileLength(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        printf("Errore nell'apertura del file %s\n", filename);
        exit(1);
    }

    uint8_t header[44];
    fread(header, sizeof(uint8_t), 44, file);
    int dataSize = header[40] | (header[41] << 8) | (header[42] << 16) | (header[43] << 24);
    fclose(file);
    return dataSize / sizeof(int16_t);
}

// Funzione per leggere i campioni audio da un file .wav
void readWavFile(const char *filename, float *x, int N) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        printf("Errore nell'apertura del file %s\n", filename);
        exit(1);
    }

    fseek(file, 44, SEEK_SET);
    int16_t *buffer = (int16_t *)malloc(N * sizeof(int16_t));
    fread(buffer, sizeof(int16_t), N, file);
    for (int i = 0; i < N; i++) {
        x[i] = (float)buffer[i];
    }

    free(buffer);
    fclose(file);
}

// Funzione per scrivere un file .wav con l'intestazione
void writeWavFile(const char *filename, float *x, int N) {
    FILE *file = fopen(filename, "wb");
    if (file == NULL) {
        printf("Errore nell'apertura del file %s\n", filename);
        exit(1);
    }

    uint8_t header[44] = {
        'R', 'I', 'F', 'F',
        0, 0, 0, 0,
        'W', 'A', 'V', 'E',
        'f', 'm', 't', ' ',
        16, 0, 0, 0,
        1, 0,
        1, 0,
        0x44, 0xAC, 0x00, 0x00,
        0x88, 0x58, 0x01, 0x00,
        2, 0,
        16, 0,
        'd', 'a', 't', 'a',
        0, 0, 0, 0
    };

    int dataSize = N * sizeof(int16_t);
    int fileSize = 44 + dataSize - 8;
    header[4] = (fileSize & 0xFF);
    header[5] = ((fileSize >> 8) & 0xFF);
    header[6] = ((fileSize >> 16) & 0xFF);
    header[7] = ((fileSize >> 24) & 0xFF);
    header[40] = (dataSize & 0xFF);
    header[41] = ((dataSize >> 8) & 0xFF);
    header[42] = ((dataSize >> 16) & 0xFF);
    header[43] = ((dataSize >> 24) & 0xFF);

    fwrite(header, sizeof(uint8_t), 44, file);

    int16_t *buffer = (int16_t *)malloc(N * sizeof(int16_t));
    for (int i = 0; i < N; i++) {
        buffer[i] = (int16_t)x[i];
    }
    fwrite(buffer, sizeof(int16_t), N, file);

    free(buffer);
    fclose(file);
}

// Kernel per la trasformata discreta di Fourier (DFT) con shared memory
__global__ void dftKernel(const float *x, Complesso *X, int N) {
    __shared__ float shared_x[BLOCK_SIZE]; // Memoria condivisa per i campioni audio

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    int globalIdx;

    float angle, cosAngle, sinAngle;
    float angleFactor = 2.0f * PI * i / N;
    float real = 0;
    float imag = 0;

    for (int blockOffset = 0; blockOffset < N; blockOffset += blockDim.x) {
        // Copia i dati nella memoria condivisa (shared memory)
        int idx = blockOffset + tid;

        if (idx < N) {
            shared_x[tid] = x[idx];
        } else {
            shared_x[tid] = 0.0f; // Padding per i thread fuori dai limiti
        }
        __syncthreads(); // Sincronizzazione tra i thread del blocco

        // Calcolo della DFT usando i dati nella shared memory
        for (int j = 0; j < blockDim.x; j+=10) {

            globalIdx = blockOffset + j;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                real = fmaf(shared_x[j], cosAngle, real);
                imag = fmaf(-shared_x[j], sinAngle, imag);
            }
            
            globalIdx = blockOffset + j+1;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                real = fmaf(shared_x[j+1], cosAngle, real);
                imag = fmaf(-shared_x[j+1], sinAngle, imag);
            }
            
            globalIdx = blockOffset + j+2;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                real = fmaf(shared_x[j+2], cosAngle, real);
                imag = fmaf(-shared_x[j+2], sinAngle, imag);
            }
            
            globalIdx = blockOffset + j+3;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                real = fmaf(shared_x[j+3], cosAngle, real);
                imag = fmaf(-shared_x[j+3], sinAngle, imag);
            }
            
            globalIdx = blockOffset + j+4;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                real = fmaf(shared_x[j+4], cosAngle, real);
                imag = fmaf(-shared_x[j+4], sinAngle, imag);
            }
            
            globalIdx = blockOffset + j+5;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                real = fmaf(shared_x[j+5], cosAngle, real);
                imag = fmaf(-shared_x[j+5], sinAngle, imag);
            }
            
            globalIdx = blockOffset + j+6;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                real = fmaf(shared_x[j+6], cosAngle, real);
                imag = fmaf(-shared_x[j+6], sinAngle, imag);
            }
            
            globalIdx = blockOffset + j+7;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                real = fmaf(shared_x[j+7], cosAngle, real);
                imag = fmaf(-shared_x[j+7], sinAngle, imag);
            }
            
            globalIdx = blockOffset + j+8;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                real = fmaf(shared_x[j+8], cosAngle, real);
                imag = fmaf(-shared_x[j+8], sinAngle, imag);
            }
            
            globalIdx = blockOffset + j+9;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                real = fmaf(shared_x[j+9], cosAngle, real);
                imag = fmaf(-shared_x[j+9], sinAngle, imag);
            }
        }
        __syncthreads(); // Assicura che tutti i thread abbiano completato il ciclo
    }

    // Scrittura del risultato nella memoria globale
    if (i < N) {
        X[i].real = real;
        X[i].imag = imag;
    }
}

// Kernel per applicare un filtro passa-basso
__global__ void filtro(Complesso *X, int N, int fc, int fs) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k < N) {
        int cutoffIndex = (fc * N) / fs;
        if (k > cutoffIndex && k < N - cutoffIndex) {
            X[k].real = 0;
            X[k].imag = 0;
        }
    }
}


// Kernel per la trasformata inversa discreta di Fourier (IDFT) con shared memory
__global__ void idftKernel(const Complesso *X, float *x, int N) {
    __shared__ Complesso shared_X[BLOCK_SIZE]; // Memoria condivisa per i valori complessi

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    int globalIdx;

    float angle, cosAngle, sinAngle;
    float angleFactor = 2.0f * PI * i / N;
    float temp = 0.0f;

    for (int blockOffset = 0; blockOffset < N; blockOffset += blockDim.x) {
        // Copia i dati nella memoria condivisa
        int idx = blockOffset + tid;
        if (idx < N) {
            shared_X[tid] = X[idx];
        } else {
            shared_X[tid].real = 0.0f;
            shared_X[tid].imag = 0.0f;
        }
        __syncthreads(); // Sincronizzazione tra i thread del blocco

        // Calcolo della IDFT usando i dati nella shared memory
        for (int j = 0; j < blockDim.x; j+=10) {

            globalIdx = blockOffset + j;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                temp = fmaf(shared_X[j].real, cosAngle, temp);
                temp = fmaf(-shared_X[j].imag, sinAngle, temp);
            }
            
            globalIdx = blockOffset + j+1;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                temp = fmaf(shared_X[j+1].real, cosAngle, temp);
                temp = fmaf(-shared_X[j+1].imag, sinAngle, temp);
            }
            
            globalIdx = blockOffset + j+2;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                temp = fmaf(shared_X[j+2].real, cosAngle, temp);
                temp = fmaf(-shared_X[j+2].imag, sinAngle, temp);
            }
            
            globalIdx = blockOffset + j+3;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                temp = fmaf(shared_X[j+3].real, cosAngle, temp);
                temp = fmaf(-shared_X[j+3].imag, sinAngle, temp);
            }
            
            globalIdx = blockOffset + j+4;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                temp = fmaf(shared_X[j+4].real, cosAngle, temp);
                temp = fmaf(-shared_X[j+4].imag, sinAngle, temp);
            }
            
            globalIdx = blockOffset + j+5;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                temp = fmaf(shared_X[j+5].real, cosAngle, temp);
                temp = fmaf(-shared_X[j+5].imag, sinAngle, temp);
            }
            
            globalIdx = blockOffset + j+6;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                temp = fmaf(shared_X[j+6].real, cosAngle, temp);
                temp = fmaf(-shared_X[j+6].imag, sinAngle, temp);
            }
            
            globalIdx = blockOffset + j+7;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                temp = fmaf(shared_X[j+7].real, cosAngle, temp);
                temp = fmaf(-shared_X[j+7].imag, sinAngle, temp);
            }
            
            globalIdx = blockOffset + j+8;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                temp = fmaf(shared_X[j+8].real, cosAngle, temp);
                temp = fmaf(-shared_X[j+8].imag, sinAngle, temp);
            }
            
            globalIdx = blockOffset + j+9;
            if (globalIdx < N) {
                angle = angleFactor * globalIdx;
                __sincosf(angle, &sinAngle, &cosAngle);
                temp = fmaf(shared_X[j+9].real, cosAngle, temp);
                temp = fmaf(-shared_X[j+9].imag, sinAngle, temp);
            }
        }
        __syncthreads(); // Assicura che tutti i thread abbiano completato il ciclo
    }
    // Scrittura del risultato nella memoria globale
    if (i < N) {
        x[i] = temp / N;
    }
}

// Funzione per scrivere un report dei tempi di esecuzione
void writeReport(const char *filename, double dftTime, double filterTime, double idftTime, double totalTime) {
    FILE *file = fopen(filename, "w");
    if (file == NULL) {
        printf("Errore nell'apertura del file %s\n", filename);
        exit(1);
    }

    fprintf(file, "Report tempi di esecuzione:\n");
    fprintf(file, "------------------------------------\n");
    fprintf(file, "DFT  : %f secondi\n", dftTime);
    fprintf(file, "Filtro: %f secondi\n", filterTime);
    fprintf(file, "IDFT : %f secondi\n", idftTime);
    fprintf(file, "Totale: %f secondi\n", totalTime);
    fprintf(file, "------------------------------------\n");
    fclose(file);
}

//main
int main(int argc, char *argv[]){
    float *x, *y;
    Complesso *X;
    int N;
    clock_t start, stop;
    double dftTime, filterTime, idftTime, totalTime;
    char *filename;

    if(argc != 2){
        printf("Utilizzo: %s <file audio.wav>\n", argv[0]);
        exit(1);
    }

    filename = argv[1];

    // Creazione delle directory ./output e ./reports se non esistono
    mkdir("./output", 0777);
    mkdir("./reports", 0777);

    N = getWavFileLength(filename);

    x = (float *)malloc(N * sizeof(float));
    y = (float *)malloc(N * sizeof(float));
    X = (Complesso *)malloc(N * sizeof(Complesso));

    char timestamp[20];
    createTimestamp(timestamp, sizeof(timestamp));

    char outputFile[256], reportFile[256];

    snprintf(outputFile, sizeof(outputFile), "./output/output_Parallelo_v5.1C_%s.wav", timestamp);
    snprintf(reportFile, sizeof(reportFile), "./reports/report_Parallelo_v5.1C_%s.txt", timestamp);

    readWavFile(filename, x, N);

    float *d_x, *d_y;
    Complesso *d_X;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_X, N * sizeof(Complesso));

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Calcolo della memoria condivisa necessaria
    size_t sharedMemorySizeDFT = blockSize * sizeof(float); // Per il kernel DFT
    size_t sharedMemorySizeIDFT = blockSize * sizeof(Complesso); // Per il kernel IDFT

    start = clock();
    dftKernel<<<gridSize, blockSize, sharedMemorySizeDFT>>>(d_x, d_X, N);
    hipDeviceSynchronize();
    stop = clock();
    dftTime = (double)(stop - start) / CLOCKS_PER_SEC;

    start = clock();
    filtro<<<gridSize, blockSize>>>(d_X, N, 1000, 44100); // Non usa shared memory
    hipDeviceSynchronize();
    stop = clock();
    filterTime = (double)(stop - start) / CLOCKS_PER_SEC;

    start = clock();
    idftKernel<<<gridSize, blockSize, sharedMemorySizeIDFT>>>(d_X, d_y, N);
    hipDeviceSynchronize();
    stop = clock();
    idftTime = (double)(stop - start) / CLOCKS_PER_SEC;

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    writeWavFile(outputFile, y, N);

    totalTime = dftTime + filterTime + idftTime;

    writeReport(reportFile, dftTime, filterTime, idftTime, totalTime);

    free(x);
    free(y);
    free(X);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_X);

    return 0;
}